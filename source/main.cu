#include "hip/hip_runtime.h"
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>

#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

using namespace cv;

// // https://gist.github.com/jefflarkin/5390993
// // Macro for checking cuda errors following a cuda launch or api call
// #define cudaCheckError()                                                \
//     {                                                                    \
//       hipError_t e = hipGetLastError();                               \
//       if (e != hipSuccess) {                                           \
//           printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__,      \
//                  hipGetErrorString(e));                                \
//           exit(0);                                                      \
//       }                                                                 \
//   }

// __global__ void imgProcessingKernel(unsigned char *d_origImg,
//                                     unsigned char *d_newImg) {
//     // each thread will work on pixel value of the image, a pixel is represented
//     // with 3 values R, G, and B
//     int col = (blockIdx.x * blockDim.x) + threadIdx.x;
//     int row = (blockIdx.y * blockDim.y) + threadIdx.y;

//     if (row == 0 || row == 255 || col == 0 || col == 767) {
//         return;
//     }

//     // gaussian blur kernel
//     int blurKernel[3][3] = {{1, 2, 1}, {2, 4, 2}, {1, 2, 1}};

//     // edge detection kernel
//     int edgeDetectionKernel[3][3] = {{1, 0, -1}, {0, 0, 0}, {-1, 0, 1}};

//     // emboss kernel
//     int embossKernel[3][3] = {{-2, -1, 0}, {-1, 1, 1}, {0, 1, 2}};

//     // matrix to hold neighbor values
//     int mat[3][3];

//     // calculate neighbor values and put them in a matrix
//     mat[0][0] = d_origImg[(col - 3) + 768 * (row - 1)];
//     mat[1][0] = d_origImg[col + 768 * (row - 1)];
//     mat[2][0] = d_origImg[(col + 3) + 768 * (row - 1)];
//     mat[0][1] = d_origImg[(col - 3) + 768 * row];
//     mat[1][1] = d_origImg[col + 768 * row];
//     mat[2][1] = d_origImg[(col + 3) + 768 * row];
//     mat[0][2] = d_origImg[(col - 3) + 768 * (row + 1)];
//     mat[1][2] = d_origImg[col + 768 * (row + 1)];
//     mat[2][2] = d_origImg[(col + 3) + 768 * (row + 1)];

//     int newRGBValue = (mat[0][0] * edgeDetectionKernel[0][0]) +
//         (mat[1][0] * edgeDetectionKernel[1][0]) +
//         (mat[2][0] * edgeDetectionKernel[2][0]) +
//         (mat[0][1] * edgeDetectionKernel[0][1]) +
//         (mat[1][1] * edgeDetectionKernel[1][1]) +
//         (mat[2][1] * edgeDetectionKernel[2][1]) +
//         (mat[0][2] * edgeDetectionKernel[0][2]) +
//         (mat[1][2] * edgeDetectionKernel[1][2]) +
//         (mat[2][2] * edgeDetectionKernel[2][2]);

//     d_newImg[col + 768 * row] = newRGBValue; // r, g, or b value is copied
// }

__host__ void imgProcessing(const Mat &h_origImg, const Mat &h_newImg) {
    std::vector<uchar> d_origImg;
    std::vector<uchar> d_newImg;
    size_t imageSize = h_origImg.rows * h_origImg.cols * sizeof(uchar);

    // copy Mat data into vectors of unsigned chars
    d_origImg.assign(h_origImg.data, h_origImg.data + h_origImg.total() * h_origImg.channels());
    d_newImg.assign(h_newImg.data, h_newImg.data + h_newImg.total() * h_newImg.channels());

    // allocate memory for the original image, new image, and convolution kernel
    // on the device
    hipMalloc((void **)d_origImg.data(), imageSize);
    hipMalloc((void **)d_newImg.data(), imageSize);

    // // copy the original image data from the host to the original image data
    // // allocated on the device
    // hipMemcpy(d_origImg, h_origImg, imgSize, hipMemcpyHostToDevice);

    // // 8 x 8 is 64 threads per block
    // dim3 threadsPerBlock(8, 8);
    // // 96 x 32 blocks or 3,072 blocks
    // dim3 numBlocks(768 / threadsPerBlock.x, 256 / threadsPerBlock.y);

    // // perform image processing with 196,608 threads total, which is enough for a
    // // 768 x 256 array
    // imgProcessingKernel<<<numBlocks, threadsPerBlock>>>(d_origImg, d_newImg);
    // hipDeviceSynchronize();
    // cudaCheckError();

    // // copy device image to host image
    // hipMemcpy(h_newImg, d_newImg, imgSize, hipMemcpyDeviceToHost);

    // hipFree(d_newImg);
}

int main(int argc, char ** argv) {

    std::string imagePath, newImagePath;

    if(argc < 3) {
        std::cout << "Not enough arguments provided. Please provide the path to a image and a path for the newly created image.\n";
        return 1;
    } else {
        imagePath = argv[1];
        newImagePath = argv[2];
    }

    // allocate memory for original image on host
    Mat h_origImg = imread(imagePath, IMREAD_COLOR);
    if(h_origImg.empty()) {
        std::cout << "Could not read the image: " << imagePath << std::endl;
        return 1;
    }

    // calculate the image size
    Size imageSize = h_origImg.size();
    int imgChannels = h_origImg.channels();
    int imgWidth = imageSize.width;
    int imgHeight = imageSize.height;

    std::cout << "Loaded an image with a width of " << imgWidth << " and a height of "
              << imgHeight << ". The image has " << imgChannels << " channels.\n";

    // create a new Mat for the processed image
    Mat h_newImg(imageSize, h_origImg.type());

    // // host function to start the image processing
    // imgProcessing(h_origImg, h_newImg, imgSize);
    // hipDeviceSynchronize();

    // create the new image
    imwrite(newImagePath, h_newImg);

    // stbi_image_free(h_origImg);
    // free(h_newImg);

    return 0;
}
