#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#define STB_IMAGE_IMPLEMENTATION
#include "../include/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../include/stb_image_write.h"

__global__ void imgProcessingKernel(unsigned char *d_origImg,
                                    unsigned char *d_newImg) {
  // each thread will work on a R, G, or B value of the image
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  d_newImg[i] = d_origImg[i];
}

__host__ void imgProcessing(unsigned char *h_origImg, unsigned char *h_newImg,
                            int imgSize) {
  unsigned char *d_origImg;
  unsigned char *d_newImg;

  // allocate memory for the original image and new image on the device
  hipMalloc((void **)&d_origImg, imgSize);
  hipMalloc((void **)&d_newImg, imgSize);

  // copy the original image data from the host to the original image data
  // allocated on the device
  hipMemcpy(d_origImg, h_origImg, imgSize, hipMemcpyHostToDevice);

  // max amount of threads in a block is 1024
  dim3 threadsPerBlock(1024);
  // calculate the amount of blocks needed
  dim3 numBlocks(imgSize / 1024);
  // perform image processing
  imgProcessingKernel<<<numBlocks, threadsPerBlock>>>(d_origImg, d_newImg);
  hipDeviceSynchronize();
  // copy device image to host image
  hipMemcpy(h_newImg, d_newImg, imgSize, hipMemcpyDeviceToHost);

  hipFree(d_newImg);
}

const char *IMG_PATH = "images/shell.jpg";

int main() {

  int imgWidth, imgHeight, imgChannels;

  // allocate memory for original image on host
  unsigned char *h_origImg =
      stbi_load(IMG_PATH, &imgWidth, &imgHeight, &imgChannels, 0);
  if (h_origImg == NULL) {
    printf("Error in loading the image\n");
    exit(1);
  }

  // calculate the image size
  size_t imgSize =
      imgChannels * imgHeight * imgChannels * sizeof(unsigned char);

  printf("Loaded an image with a width of %dpx, a height of %dpx and %d "
         "channels\n",
         imgWidth, imgHeight, imgChannels);

  // allocate memory for the new image on host
  unsigned char *h_newImg = (unsigned char *)malloc(imgSize);

  // host function to start the image processing
  imgProcessing(h_origImg, h_newImg, imgSize);
  hipDeviceSynchronize();

  // create the new image
  stbi_write_jpg("images/shell-copy.jpg", imgWidth, imgHeight, imgChannels,
                 h_newImg, 100);

  stbi_image_free(h_origImg);
  free(h_newImg);

  return 0;
}